// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)
#include <gtsam_points/cuda/cuda_memory.hpp>

#include <hip/hip_runtime.h>
#include <gtsam_points/cuda/check_error.cuh>

namespace gtsam_points {

void *cuda_malloc(size_t size, ihipStream_t *stream) {
  void *ptr = nullptr;
#if CUDAToolkit_VERSION_MAJOR == 11
#if CUDAToolkit_VERSION_MINOR >= 4
  check_error << hipMallocAsync(&ptr, size, stream);
#else
  check_error << hipMalloc(&ptr, size);
#endif
#endif
  return ptr;
}

void *cuda_malloc_and_upload(const void *data, size_t size,
                             ihipStream_t *stream) {
  void *ptr = nullptr;

#if CUDAToolkit_VERSION_MAJOR == 11
#if CUDAToolkit_VERSION_MINOR >= 4
  check_error << hipMallocAsync(&ptr, size, stream);
#else
  check_error << hipMalloc(&ptr, size);
#endif
#endif

  check_error << hipMemcpyAsync(ptr, data, size, hipMemcpyHostToDevice,
                                 stream);
  return ptr;
}

void cuda_free(void *ptr, ihipStream_t *stream) {
#if CUDAToolkit_VERSION_MAJOR == 11
#if CUDAToolkit_VERSION_MINOR >= 4
  check_error << hipFreeAsync(ptr, stream);
#else
  check_error << hipFree(ptr);
#endif
#endif
}

void cuda_host_to_device(void *dst, const void *src, size_t size,
                         ihipStream_t *stream) {
  check_error << hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice,
                                 stream);
}

void cuda_device_to_host(void *dst, const void *src, size_t size,
                         ihipStream_t *stream) {
  check_error << hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost,
                                 stream);
}

void cuda_mem_get_info(size_t *free, size_t *total) {
  check_error << hipMemGetInfo(free, total);
}

} // namespace gtsam_points
